#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>
#include ""

using namespace std;

bool readFile(std::string filename, char** data, int* rows, int* columns);
__device__ bool checkForPattern(char* input, char* pattern, int inputRows, int inputColumns, int patternRows, int patternColumns, int i, int j);

// CUDA kernel for pattern matching
__global__ void patternMatchingKernel(char* input, int inputRows, int inputColumns, char* pattern, int patternRows, int patternColumns, int* resultCoords, int maxCoordsSize) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    int i = k / inputColumns;
    int j = k % inputColumns;

    // Perform pattern matching using the checkForPattern function
    if (i < inputRows && j < inputColumns)
    {
        bool patternFound = checkForPattern(input, pattern, inputRows, inputColumns, patternRows, patternColumns, i, j);

        // Calculate index for resultCoords array
        int index = i * inputColumns + j;
        if (patternFound)
        {
            resultCoords[index * 2] = i;
            resultCoords[index * 2 + 1] = j;
        }
        else
        {
            resultCoords[index * 2] = -1;
            resultCoords[index * 2 + 1] = -1;
        }
    }
}

int main(int argc, char* argv[]) {

    std::string inputFilename, patternFilename;

    // C++ makes the first argument the name of the program, so two additional arguments make 3
    if (argc != 3)
    {
        cout << "Program accepts two arguments: <input filename> and <pattern filename>" << endl;
        return 1;
    }

    inputFilename = argv[1];
    patternFilename = argv[2];

    // read in both files
    char* input = nullptr;
    char* pattern = nullptr;
    int inputRows, inputColumns, patternRows, patternColumns;

    if (!readFile(inputFilename, &input, &inputRows, &inputColumns) ||
        !readFile(patternFilename, &pattern, &patternRows, &patternColumns))
    {
        return 1;
    }

    for (int i = 0; i < inputColumns * inputRows; i++)
    {
        cout << input[i];
    }

    int maxCoordsSize = inputRows * inputColumns * 2;

    // Allocate device memory for input, pattern, and result coordinates
    char* d_input;
    char* d_pattern;
    int* d_resultCoords;

    hipMalloc(&d_input, inputRows * inputColumns * sizeof(char));
    hipMalloc(&d_pattern, patternRows * patternColumns * sizeof(char));
    hipMalloc(&d_resultCoords, maxCoordsSize * sizeof(int));

    // Copy input and pattern data from host to device
    hipMemcpy(d_input, input, inputRows * inputColumns * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_pattern, pattern, patternRows * patternColumns * sizeof(char), hipMemcpyHostToDevice);

    // Launch CUDA kernel for pattern matching
    patternMatchingKernel << < 1, 32 >> > (d_input, inputRows, inputColumns, d_pattern, patternRows, patternColumns, d_resultCoords, maxCoordsSize);

    // Copy result coordinates from device to host
    int* resultCoords = new int[maxCoordsSize];
    hipMemcpy(resultCoords, d_resultCoords, maxCoordsSize * sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_input);
    hipFree(d_pattern);
    hipFree(d_resultCoords);

    for (int i = 0; i < maxCoordsSize; i += 2)
    {
        //if (resultCoords[i] >= 0)
        cout << resultCoords[i] << ", " << resultCoords[i + 1] << endl;
    }

    delete[] resultCoords;

    return 0;
}

bool readFile(std::string filename, char** data, int* rows, int* columns)
{
    std::ifstream file(filename);

    *rows = 0;
    *columns = 0;

    //Open the file and read line by line into array
    if (file.is_open())
    {
        // Get the total number of lines in the file by counting the number of newline characters
        int totalLines = std::count(std::istreambuf_iterator<char>(file),
            std::istreambuf_iterator<char>(), '\n') + 1;

        // Initiailize the array with the total number of lines
        char** arr = new char* [totalLines];
        *rows = totalLines;

        //Go back to the start of the file
        file.clear();
        file.seekg(0, std::ios::beg);

        for (int i = 0; i < totalLines; i++)
        {
            std::string line;
            if (!std::getline(file, line, '\n'))
            {
                break; // Break out of the loop on error
            }

            *columns = line.length();
            arr[i] = new char[line.length()];

            for (int j = 0; j < line.length(); j++)
            {
                arr[i][j] = line[j];
            }
        }

        char* flat = new char[(*rows) * (*columns)];
        for (int i = 0; i < *rows; i++)
        {
            for (int j = 0; j < *columns; j++)
            {
                flat[i * (*columns) + j] = arr[i][j];
            }
        }

        *data = flat;

        return true;
    }
    else
    {
        std::cout << filename << " not opened" << std::endl;
        return false;
    }
}

/*
Checks for the pattern within the input file
Ignores wildcard character '*' and returns true or false if pattern is found or not
*/
__device__ bool checkForPattern(char* input, char* pattern, int inputRows, int inputColumns, int patternRows, int patternColumns, int i, int j)
{
    bool patternFound = true;

    // loop through each character in the pattern, stop as soon as pattern is out of bounds or it doesn't match
    for (int currentRow = 0; currentRow < patternRows && patternFound; currentRow++)
    {
        for (int currentColumn = 0; currentColumn < patternColumns && patternFound; currentColumn++)
        {
            if (pattern[currentRow * patternColumns + currentColumn] != '*' &&
                (i + currentRow < 0 || i + currentRow >= inputRows || j + currentColumn < 0 || j + currentColumn >= inputColumns || // out of bounds
                    input[(i + currentRow) * inputColumns + (j + currentColumn)] != pattern[currentRow * patternColumns + currentColumn])) // doesn't match pattern
            {
                patternFound = false;
            }
        }
    }

    return patternFound;
}