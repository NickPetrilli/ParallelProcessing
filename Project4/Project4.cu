#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>
#include ""

using namespace std;

bool readFile(std::string filename, char** data, int* rows, int* columns);
__device__ bool checkForPattern(char* input, char* pattern, int inputRows, int inputColumns, int patternRows, int patternColumns, int i, int j);

// CUDA kernel for pattern matching
__global__ void patternMatchingKernel(char* input, int inputRows, int inputColumns, char* pattern, int patternRows, int patternColumns, int* resultCoords, int maxCoordsSize) {
    int stride = blockDim.x * gridDim.x;
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    while (k < inputRows * inputColumns)
    {
        int i = k / inputColumns;
        int j = k % inputColumns;

        // Perform pattern matching using the checkForPattern function
        bool patternFound = checkForPattern(input, pattern, inputRows, inputColumns, patternRows, patternColumns, i, j);

        // Calculate index for resultCoords array
        int index = i * inputColumns + j;
        if (patternFound)
        {
            resultCoords[index * 2] = i;
            resultCoords[index * 2 + 1] = j;
        }
        else
        {
            resultCoords[index * 2] = -1;
            resultCoords[index * 2 + 1] = -1;
        }

        k += stride;
    }
}

int main(int argc, char* argv[]) {

    std::string inputFilename, patternFilename;
    int numBlocks = 1, numThreads = 1;

    // C++ makes the first argument the name of the program, so 4 additional arguments make 5
    if (argc != 5)
    {
        cout << "Program accepts four arguments: <input filename>, <pattern filename>, <num blocks>, <num threads>" << endl;
        return 1;
    }

    inputFilename = argv[1];
    patternFilename = argv[2];

    try
    {
        numBlocks = stoi(argv[3]);
        numThreads = stoi(argv[4]);
    }
    catch (...)
    {
        cout << "Must provide integers for the number of blocks and threads" << endl;
        return 1;
    }

    // read in both files
    char* input = nullptr;
    char* pattern = nullptr;
    int inputRows, inputColumns, patternRows, patternColumns;

    if (!readFile(inputFilename, &input, &inputRows, &inputColumns) ||
        !readFile(patternFilename, &pattern, &patternRows, &patternColumns))
    {
        cout << "Failed to read files" << endl;
        return 1;
    }

    // Print arguments
    cout << "Input File: " << inputFilename << endl;
    cout << "Patter File: " << patternFilename << endl;
    cout << "Number of Blocks: " << numBlocks << endl;
    cout << "Number of Threads: " << numThreads << endl;


    // Allocate device memory for input, pattern, and result coordinates
    int maxCoordsSize = inputRows * inputColumns * 2;
    char* d_input;
    char* d_pattern;
    int* d_resultCoords;

    hipMalloc(&d_input, inputRows * inputColumns * sizeof(char));
    hipMalloc(&d_pattern, patternRows * patternColumns * sizeof(char));
    hipMalloc(&d_resultCoords, maxCoordsSize * sizeof(int));

    // Copy input and pattern data from host to device
    hipMemcpy(d_input, input, inputRows * inputColumns * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_pattern, pattern, patternRows * patternColumns * sizeof(char), hipMemcpyHostToDevice);

    // Launch CUDA kernel for pattern matching
    patternMatchingKernel << < numBlocks, numThreads >> > (d_input, inputRows, inputColumns, d_pattern, patternRows, patternColumns, d_resultCoords, maxCoordsSize);

    // Copy result coordinates from device to host
    int* resultCoords = new int[maxCoordsSize];
    hipMemcpy(resultCoords, d_resultCoords, maxCoordsSize * sizeof(int), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_input);
    hipFree(d_pattern);
    hipFree(d_resultCoords);

    // Print results (to console for now)
    for (int i = 0; i < maxCoordsSize; i += 2)
    {
        if (resultCoords[i] >= 0)
            cout << resultCoords[i] << ", " << resultCoords[i + 1] << endl;
    }

    delete[] resultCoords;

    return 0;
}

bool readFile(std::string filename, char** data, int* rows, int* columns)
{
    std::ifstream file(filename);

    *rows = 0;
    *columns = 0;

    //Open the file and read line by line into array
    if (file.is_open())
    {
        // Get the total number of lines in the file by counting the number of newline characters
        int totalLines = std::count(std::istreambuf_iterator<char>(file),
            std::istreambuf_iterator<char>(), '\n') + 1;

        // Initiailize the array with the total number of lines
        char** arr = new char* [totalLines];
        *rows = totalLines;

        //Go back to the start of the file
        file.clear();
        file.seekg(0, std::ios::beg);

        for (int i = 0; i < totalLines; i++)
        {
            std::string line;
            if (!std::getline(file, line, '\n'))
            {
                break; // Break out of the loop on error
            }

            *columns = line.length();
            arr[i] = new char[line.length()];

            for (int j = 0; j < line.length(); j++)
            {
                arr[i][j] = line[j];
            }
        }

        char* flat = new char[(*rows) * (*columns)];
        for (int i = 0; i < *rows; i++)
        {
            for (int j = 0; j < *columns; j++)
            {
                flat[i * (*columns) + j] = arr[i][j];
            }
        }

        *data = flat;

        return true;
    }
    else
    {
        std::cout << filename << " not opened" << std::endl;
        return false;
    }
}

/*
Checks for the pattern within the input file
Ignores wildcard character '*' and returns true or false if pattern is found or not
*/
__device__ bool checkForPattern(char* input, char* pattern, int inputRows, int inputColumns, int patternRows, int patternColumns, int i, int j)
{
    bool patternFound = true;

    // loop through each character in the pattern, stop as soon as pattern is out of bounds or it doesn't match
    for (int currentRow = 0; currentRow < patternRows && patternFound; currentRow++)
    {
        for (int currentColumn = 0; currentColumn < patternColumns && patternFound; currentColumn++)
        {
            if (pattern[currentRow * patternColumns + currentColumn] != '*' &&
                (i + currentRow < 0 || i + currentRow >= inputRows || j + currentColumn < 0 || j + currentColumn >= inputColumns || // out of bounds
                    input[(i + currentRow) * inputColumns + (j + currentColumn)] != pattern[currentRow * patternColumns + currentColumn])) // doesn't match pattern
            {
                patternFound = false;
            }
        }
    }

    return patternFound;
}